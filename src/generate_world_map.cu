#include "hip/hip_runtime.h"
/**
 * RTX 2080 Ti: CUDA Cores 4352, compute capablity: 7.5
 * 68 SMs, 64 CUDA Cores/SM
 * Block size: 32 x 32 = 1024 threads
 * mean_x, mean_y, sigma, scale, rho: 5 * # of distributions
 * resolution, size: 2
 * map: pWorldMapSize * pWorldMapSize
 **/

#include <CoverageControl/generate_world_map.ch>
#include <hip/hip_runtime.h>
#include <cmath>
#include <cuda_helpers/hip/hip_runtime_api.h>
#include <thrust/extrema.h>
#include <thrust/device_ptr.h>

__device__ __constant__ int cu_num_dists;
__device__ __constant__ int cu_map_size;
__device__ __constant__ float cu_resolution;
__device__ __constant__ float cu_truncation;
__device__ __constant__ float cu_OneBySqrt2;
__device__ __constant__ float cu_normalization_factor;

__device__
float2 TransformPoint(BND_Cuda const *device_dists, int i, float2 const &in_point) {
	float2 pt;
	auto bnd = device_dists[i];
	if(bnd.rho == 0) {
		pt.x = (in_point.x - bnd.mean_x)/bnd.sigma_x;
		pt.y = (in_point.y - bnd.mean_y)/bnd.sigma_y;
		return pt;
	}
	pt.x = (in_point.x - bnd.mean_x)/bnd.sigma_x;
	pt.y = (in_point.y - bnd.mean_y)/bnd.sigma_y;
	pt.x = (pt.x - bnd.rho * pt.y)/(sqrt(1 - bnd.rho*bnd.rho));
	return pt;
}

__device__
float IntegrateQuarterPlane (BND_Cuda const &bnd, float2 const &in_point) {
	float2 pt;
	if(bnd.rho == 0) {
		pt.x = (in_point.x - bnd.mean_x)/bnd.sigma_x;
		pt.y = (in_point.y - bnd.mean_y)/bnd.sigma_y;
	} else {
		pt.x = (in_point.x - bnd.mean_x)/bnd.sigma_x;
		pt.y = (in_point.y - bnd.mean_y)/bnd.sigma_y;
		pt.x = (pt.x - bnd.rho * pt.y)/(sqrt(1 - bnd.rho*bnd.rho));
	}
	/* auto transformed_point = TransformPoint(i, in_point); */
	float sc = bnd.scale;
	/* return sc; */
	return sc * erfc(pt.x / sqrt(2.)) * erfc(pt.y / sqrt(2.))/4.;
}

__device__
float ComputeImportanceRectangle (BND_Cuda const *device_dists, float2 const &bottom_left, float2 const &top_right) {
	float2 bottom_right = make_float2(top_right.x, bottom_left.y);
	float2 top_left = make_float2(bottom_left.x, top_right.y);

	float total_importance = 0;
	for(int i = 0; i < cu_num_dists; ++i) {
		float2 mid_pt = make_float2((bottom_left.x + top_right.x)/2., (bottom_left.y + top_right.y)/2.);
		auto bnd = device_dists[i];
		if(bnd.rho == 0) {
			mid_pt.x = (mid_pt.x - bnd.mean_x)/bnd.sigma_x;
			mid_pt.y = (mid_pt.y - bnd.mean_y)/bnd.sigma_y;
		} else {
			mid_pt.x = (mid_pt.x - bnd.mean_x)/bnd.sigma_x;
			mid_pt.y = (mid_pt.y - bnd.mean_y)/bnd.sigma_y;
			mid_pt.x = (mid_pt.x - bnd.rho * mid_pt.y)/(sqrt(1 - bnd.rho*bnd.rho));
		}
		if(mid_pt.x * mid_pt.x + mid_pt.y * mid_pt.y > cu_truncation * cu_truncation + cu_resolution * cu_resolution) {
			/* printf("%f, %f, %f, %f\n", mid_pt.x, mid_pt.y, cu_truncation, cu_resolution); */
			continue;
		}
		total_importance += IntegrateQuarterPlane(bnd, bottom_left);
		total_importance -= IntegrateQuarterPlane(bnd, bottom_right);
		total_importance -= IntegrateQuarterPlane(bnd, top_left);
		total_importance += IntegrateQuarterPlane(bnd, top_right);
	}
	return total_importance;
}

__global__ void kernel (BND_Cuda const *device_dists, float *importance_vec) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	int vec_idx = idx * cu_map_size + idy;
	if(not (idx < cu_map_size and idy < cu_map_size)) {
		return;
	}
	float2 bottom_left = make_float2(idx * cu_resolution, idy * cu_resolution);
	float2 top_right = make_float2(idx * cu_resolution + cu_resolution, idy * cu_resolution + cu_resolution);
	importance_vec[vec_idx] = ComputeImportanceRectangle(device_dists, bottom_left, top_right);
}

__global__ void normalize (float *importance_vec) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	int vec_idx = idx * cu_map_size + idy;
	if(not (idx < cu_map_size and idy < cu_map_size)) {
		return;
	}
	importance_vec[vec_idx] *= cu_normalization_factor;
}
void generate_world_map_cuda(BND_Cuda *host_dists, Polygons_Cuda const &host_polygons, int const num_dists, int const map_size, float const resolution, float const truncation, float const pNorm, float *host_importance_vec, float &normalization_factor) {

	checkCudaErrors(hipDeviceSynchronize());

	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(cu_num_dists), &num_dists, sizeof(int)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(cu_map_size), &map_size, sizeof(int)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(cu_resolution), &resolution, sizeof(float)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(cu_truncation), &truncation, sizeof(float)));
	float f_OneBySqrt2 = (float)(1./std::sqrt(2.));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(cu_OneBySqrt2), &f_OneBySqrt2, sizeof(float)));

	BND_Cuda *device_dists;
	checkCudaErrors(hipMalloc(&device_dists, num_dists * sizeof(BND_Cuda)));
	/* checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(device_dists), &device_dists, sizeof(device_dists))); */
	checkCudaErrors(hipMemcpy(device_dists, host_dists, num_dists * sizeof(BND_Cuda), hipMemcpyHostToDevice));

	/* Polygons_Cuda device_polygons; */
	/* checkCudaErrors(hipMalloc(&(device_polygons.x), host_polygons.num_pts * sizeof(float))); */
	/* checkCudaErrors(hipMalloc(&(device_polygons.y), host_polygons.num_pts * sizeof(float))); */
	/* checkCudaErrors(hipMalloc(&(device_polygons.sz), host_polygons.num_polygons * sizeof(int))); */
	/* device_polygons.num_pts = host_polygons.num_pts; */
	/* device_polygons.num_polygons = host_polygons.num_polygons; */

	float *device_importance_vec;
	checkCudaErrors(hipMalloc(&device_importance_vec, map_size * map_size * sizeof(float)));


	dim3 dimBlock(16, 16, 1);
	dim3 dimGrid(map_size/dimBlock.x, map_size/dimBlock.x, 1);

	kernel <<<dimGrid, dimBlock>>>(device_dists, device_importance_vec);

	thrust::device_ptr<float> d_ptr = thrust::device_pointer_cast(device_importance_vec);
	float max = *(thrust::max_element(d_ptr, d_ptr + map_size * map_size));

	normalization_factor = pNorm;
	if(max > 1e-8) {
		normalization_factor = pNorm / max;
	}
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(cu_normalization_factor), &normalization_factor, sizeof(float)));
	normalize <<<dimGrid, dimBlock>>>(device_importance_vec);

	checkCudaErrors(hipMemcpy(host_importance_vec, device_importance_vec, map_size * map_size * sizeof(float), hipMemcpyDeviceToHost));

	checkCudaErrors(hipFree(device_dists));
	checkCudaErrors(hipFree(device_importance_vec));
	/* checkCudaErrors(hipFree(device_polygons.x)); */
	/* checkCudaErrors(hipFree(device_polygons.y)); */
	/* checkCudaErrors(hipFree(device_polygons.sz)); */

	/* auto largest_val_ptr = thrust::max_element(host_importance_vec, host_importance_vec + map_size * map_size); */

	hipError_t error = hipGetLastError();
	if(error != hipSuccess) {
		std::stringstream strstr;
		strstr << "run_kernel launch failed" << std::endl;
		throw strstr.str();
	}
}
