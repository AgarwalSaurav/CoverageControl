#include "hip/hip_runtime.h"
/**
	* RTX 2080 Ti: CUDA Cores 4352, compute capablity: 7.5
	* 68 SMs, 64 CUDA Cores/SM
	* Block size: 32 x 32 = 1024 threads
	* mean_x, mean_y, sigma, scale, rho: 5 * # of distributions
	* resolution, size: 2
	* map: pWorldMapSize * pWorldMapSize
	**/

#include <CoverageControl/generate_world_map.ch>
#include <hip/hip_runtime.h>
#include <cmath>
#include <hip/hip_runtime_api.h>


__device__ __constant__ int cu_num_dists;
__device__ __constant__ int cu_map_size;
__device__ __constant__ float cu_resolution;
__device__ __constant__ float cu_truncation;
__device__ __constant__ float cu_OneBySqrt2;

__device__
float2 TransformPoint(BND_Cuda const *cu_dists, int i, float2 const &in_point) {
	float2 pt;
	auto bnd = cu_dists[i];
	if(bnd.rho == 0) {
		pt.x = (in_point.x - bnd.mean_x)/bnd.sigma_x;
		pt.y = (in_point.y - bnd.mean_y)/bnd.sigma_y;
		return pt;
	}
	pt.x = (in_point.x - bnd.mean_x)/bnd.sigma_x;
	pt.y = (in_point.y - bnd.mean_y)/bnd.sigma_y;
	pt.x = (pt.x - bnd.rho * pt.y)/(sqrt(1 - bnd.rho*bnd.rho));
	return pt;
}

__device__
float IntegrateQuarterPlane (BND_Cuda const &bnd, float2 const &in_point) {
	float2 pt;
	if(bnd.rho == 0) {
		pt.x = (in_point.x - bnd.mean_x)/bnd.sigma_x;
		pt.y = (in_point.y - bnd.mean_y)/bnd.sigma_y;
	} else {
		pt.x = (in_point.x - bnd.mean_x)/bnd.sigma_x;
		pt.y = (in_point.y - bnd.mean_y)/bnd.sigma_y;
		pt.x = (pt.x - bnd.rho * pt.y)/(sqrt(1 - bnd.rho*bnd.rho));
	}
	/* auto transformed_point = TransformPoint(i, in_point); */
	float sc = bnd.scale;
	/* return sc; */
	return sc * erfc(pt.x / sqrt(2.)) * erfc(pt.y / sqrt(2.))/4.;
}

__device__
float ComputeImportanceRectangle (BND_Cuda const *cu_dists, float2 const &bottom_left, float2 const &top_right) {
	float2 bottom_right = make_float2(top_right.x, bottom_left.y);
	float2 top_left = make_float2(bottom_left.x, top_right.y);

	float total_importance = 0;
	for(int i = 0; i < cu_num_dists; ++i) {
		float2 mid_pt = make_float2((bottom_left.x + top_right.x)/2., (bottom_left.y + top_right.y)/2.);
		auto bnd = cu_dists[i];
		if(bnd.rho == 0) {
			mid_pt.x = (mid_pt.x - bnd.mean_x)/bnd.sigma_x;
			mid_pt.y = (mid_pt.y - bnd.mean_y)/bnd.sigma_y;
		} else {
			mid_pt.x = (mid_pt.x - bnd.mean_x)/bnd.sigma_x;
			mid_pt.y = (mid_pt.y - bnd.mean_y)/bnd.sigma_y;
			mid_pt.x = (mid_pt.x - bnd.rho * mid_pt.y)/(sqrt(1 - bnd.rho*bnd.rho));
		}
		if(mid_pt.x * mid_pt.x + mid_pt.y * mid_pt.y > cu_truncation * cu_truncation + cu_resolution * cu_resolution) {
			/* printf("%f, %f, %f, %f\n", mid_pt.x, mid_pt.y, cu_truncation, cu_resolution); */
			continue;
		}
		total_importance += IntegrateQuarterPlane(bnd, bottom_left);
		total_importance -= IntegrateQuarterPlane(bnd, bottom_right);
		total_importance -= IntegrateQuarterPlane(bnd, top_left);
		total_importance += IntegrateQuarterPlane(bnd, top_right);
	}
	return total_importance;
}

__global__ void kernel (BND_Cuda const *cu_dists, float *importance_vec) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	int vec_idx = idx * cu_map_size + idy;
	if(not (idx < cu_map_size and idy < cu_map_size)) {
		return;
	}
	float2 bottom_left = make_float2(idx * cu_resolution, idy * cu_resolution);
	float2 top_right = make_float2(idx * cu_resolution + cu_resolution, idy * cu_resolution + cu_resolution);
	/* if(total_importance > 1e-5) { */
	/* 	printf("%d, %d, %d, %.5f, %.5f, %.5f\n", idx, idy, vec_idx, bottom_left.x, bottom_left.y, total_importance); */
	/* } */
	/* printf("%d, %d, %f , %f, %f\n", cu_num_dists, cu_map_size, cu_truncation, cu_resolution, cu_OneBySqrt2); */
	/* return; */
	/* printf("%f, %f, %f , %f, %f\n", cu_dists[0].mean_x, cu_dists[0].mean_y, cu_truncation, cu_resolution, cu_OneBySqrt2); */
	importance_vec[vec_idx] = ComputeImportanceRectangle(cu_dists, bottom_left, top_right);
}

void generate_world_map_cuda(BND_Cuda *host_dists, int num_dists, int map_size, float resolution, float truncation, float *host_importance_vec) {

	checkCudaErrors(hipDeviceSynchronize());

	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(cu_num_dists), &num_dists, sizeof(int)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(cu_map_size), &map_size, sizeof(int)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(cu_resolution), &resolution, sizeof(float)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(cu_truncation), &truncation, sizeof(float)));
	float f_OneBySqrt2 = (float)(1./std::sqrt(2.));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(cu_OneBySqrt2), &f_OneBySqrt2, sizeof(float)));

	BND_Cuda *cu_dists;
	checkCudaErrors(hipMalloc(&cu_dists, num_dists * sizeof(BND_Cuda)));
	/* checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(cu_dists), &device_dists, sizeof(device_dists))); */
	checkCudaErrors(hipMemcpy(cu_dists, host_dists, num_dists * sizeof(BND_Cuda), hipMemcpyHostToDevice));

	float *device_importance_vec;
	checkCudaErrors(hipMalloc(&device_importance_vec, map_size * map_size * sizeof(float)));


	dim3 dimBlock(16, 16, 1);
	dim3 dimGrid(map_size/dimBlock.x, map_size/dimBlock.x, 1);

	kernel <<<dimGrid, dimBlock>>>(cu_dists, device_importance_vec);

	checkCudaErrors(hipMemcpy(host_importance_vec, device_importance_vec, map_size * map_size * sizeof(float), hipMemcpyDeviceToHost));

	checkCudaErrors(hipFree(cu_dists));
	checkCudaErrors(hipFree(device_importance_vec));

	hipError_t error = hipGetLastError();
	if(error != hipSuccess) {
		std::stringstream strstr;
		strstr << "run_kernel launch failed" << std::endl;
		throw strstr.str();
	}
}

